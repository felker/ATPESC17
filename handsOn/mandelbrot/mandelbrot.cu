

#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>

#define MXITER 1000
#define NPOINTS 2000

typedef struct {
  
  double r;
  double i;
  
}d_complex;

// return 1 if c is outside the mandelbrot set
// return 0 if c is inside the mandelbrot set

// TASK 1: make this a device function 
int testpoint(d_complex c){
  
  d_complex z;
  
  int iter;
  double temp;
  
  z = c;
  
  for(iter=0; iter<MXITER; iter++){
    
    temp = (z.r*z.r) - (z.i*z.i) + c.r;
    
    z.i = z.r*z.i*2. + c.i;
    z.r = temp;
    
    if((z.r*z.r+z.i*z.i)>4.0){
      return 1;
    }
  }
  
  return 0;
  
}


// TASK 2: make this a kernel that processes 
// (i,j) \in   [blockIdx.x*blockDim.x,(blockIdx.x+1)*blockDim.x) 
//           x [blockIdx.y*blockDim.y,(blockIdx.y+1)*blockDim.y) 
//  and sums up the number of outside pixels in each block

// TASK 2a: annotate this to indicate it is a kernel and change return type to void
int  mandeloutside(){

  int i,j;
  double eps = 1e-5;

  d_complex c;

  int numoutside = 0;

  // TASK 2b: replace loop structures with (i,j) defined from blockIdx, blockDim, threadIdx
  for(i=0;i<NPOINTS;i++){
    for(j=0;j<NPOINTS;j++){
      c.r = -2. + 2.5*(double)(i)/(double)(NPOINTS)+eps;
      c.i =       1.125*(double)(j)/(double)(NPOINTS)+eps;

      // TASK 2c: replace this with a partial sum reduction of numoutside in thread block
      numoutside += testpoint(c);
    }
  }
  // TASK 2d: remove this
  return numoutside;
}

int main(int argc, char **argv){

  // TASK 3a: define dim3 variables for the grid size and thread-block size
  
  // TASK 3b: use cudaMalloc to create a DEVICE array that has one entry for each thread-block
  int *c_outsideCounts;

  // TASK 3c: replace this with a kernel call
  double numoutside = mandeloutside();
  
  // TASK 3d: allocate a HOST array to receive the contents of the c_outsideCounts array
  int *h_outsideCounts;

  // TASK 3e: use cudaMemcpy to copy the contents of the entries of c_outsideCounts to h_outsideCounts
  
  // TASK 3f: sum up the outsideCounts 
  int nummoutside = 0;

  double area = 2.*2.5*1.125*(NPOINTS*NPOINTS-numoutside)/(NPOINTS*NPOINTS);

  printf("area = %g\n", area);

  return 0;
}  
